//
// nvcc -arch sm_60 -O3 -o xxx hello.cu
//

#include <hip/hip_runtime.h>
#include <stdio.h>

void __global__ hello_from_gpu()
{
    printf("\n\n---Hello World from the GPU!\n\n");
}


int main(int argc, const char * argv[])
{
    printf("\n----------------------\n");
	  printf("Hello World from CPU! Before calling 'hello_from_gpu' kernel function.\n");

    hello_from_gpu<<<1, 1>>>();

	  printf("Hello World from CPU!  After calling 'hello_from_gpu' kernel function.\n");
    printf("\n----------------------\n");

    hipDeviceSynchronize(); //cudaDeviceReset();
    return 0;
}
